//4.1 MartrixMul


#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <iostream>

#define checkCudaErrors(call)                                                                 \
    do {                                                                                      \
        hipError_t err = call;                                                               \
        if (err != hipSuccess) {                                                             \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                               \
        }                                                                                     \
    } while (0)

using namespace std;

// Matrix multiplication Cuda
__global__ void matrixMultiplication(int *a, int *b, int *c, int n) {
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;

    if (row < n && col < n)
        for (int j = 0; j < n; j++) {
            sum = sum + a[row * n + j] * b[j * n + col];
        }

    c[n * row + col] = sum;
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 10;

    a = new int[n * n];
    b = new int[n * n];
    c = new int[n * n];
    int *d = new int[n * n];
    int size = n * n * sizeof(int);
    checkCudaErrors(hipMalloc(&a_dev, size));
    checkCudaErrors(hipMalloc(&b_dev, size));
    checkCudaErrors(hipMalloc(&c_dev, size));

    // Array initialization
    for (int i = 0; i < n * n; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    cout << "Given matrix A is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << a[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    cout << "Given matrix B is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << b[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    hipEvent_t start, end;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    checkCudaErrors(hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);

    // GPU Multiplication
    checkCudaErrors(hipEventRecord(start));
    matrixMultiplication<<<blocksPerGrid, threadsPerBlock>>>(a_dev, b_dev, c_dev, n);

    checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipEventSynchronize(end));

    float time = 0.0;
    checkCudaErrors(hipEventElapsedTime(&time, start, end));

    checkCudaErrors(hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost));

    // CPU matrix multiplication
    int sum = 0;
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            sum = 0;
            for (int k = 0; k < n; k++) sum = sum + a[row * n + k] * b[k * n + col];
            d[row * n + col] = sum;
        }
    }

    cout << "CPU product is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << d[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    cout << "GPU product is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << c[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    int error = 0;
    int _c, _d;
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            _c = c[row * n + col];
            _d = d[row * n + col];
            error += _c - _d;
            if (0 != (_c - _d)) {
                cout << "Error at (" << row << ", " << col << ") => GPU: " << _c << ", CPU: " << _d
                     << "\n";
            }
        }
    }
    cout << "\n";

    cout << "Error : " << error;
    cout << "\nTime Elapsed: " << time;

    return 0;
}

